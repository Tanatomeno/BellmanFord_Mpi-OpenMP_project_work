/*
 * Course: High Performance Computing 2023/2024
 *
 * Lecturer: Francesco Moscato	fmoscato@unisa.it
 *
 * Author:
 * Del Gaudio Nunzio		0612705044		n.delgaudio5@studenti.unisa.it
 *
 * Copyright (C) 2024 - All Rights Reserved
 *
 * This file is part of ProjectHPC.
 *
 * This program is free software: you can redistribute it and/or modify it under the terms of
 * the GNU General Public License as published by the Free Software Foundation, either version
 * 3 of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY;
 * without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 * See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with ContestOMP.
 * If not, see <https://www.gnu.org/licenses/gpl-3.0.html>.
 */




/*-----------------------------------------------------------------------------------------------------------------
//NOTE: Due to linking issues, all functions previously declared in the project have been redefined in this file.
//These functions have not been commented.
/-----------------------------------------------------------------------------------------------------------------*/


#include <hip/hip_runtime.h>

#include <omp.h>
#include <ctype.h>
#include <stdio.h> //printf and fprint
#include <stdlib.h> //malloc
#include <string.h> //strcpy
#include <sys/time.h> //for csv
#include <limits.h> //for INT_MAX
#include <algorithm>

#define ABSENTE_EDGE INT_MAX
#define INF INT_MAX
#define DEFAULT_FILENAME "file/RandomMatrix"
#define DEFAULT_FILENAME1D "file/RandomMatrix"
#define DEFAULT_V 50 // Default Vertexs number
#define DEFAULT_EDGE_PROBABILITY 0.8
#define DEFAULT_WEIGHT_MAX 20
#define DEFAULT_WEIGHT_MIN 1   //In order to reduce the probability of negative cycle
#define DEFAULT_SOURCE 0
#define DEFAULT_FILENAME_BellmanFord "file/Bellmanford"
#define DEFAULT_FILENAME_BellmanFord_Parallel "file/Bellmanford_Parallel"
#define DEFAULT_FILENAME_BellmanFord_Cuda "file/Bellmanford_Cuda"



//In order to avoid having to rewrite an error check for each cuda function
#define CUDA_CHECK(call) {\
    hipError_t cudaStatus = call;\
    if (cudaStatus != hipSuccess) {\
        fprintf(stderr, "CUDA_ERROR: %s in file %s at line %d\nerror number:", hipGetErrorString(cudaStatus), __FILE__, __LINE__ );\
        exit(EXIT_FAILURE);\
    }\
}


/*
for Compute Capability 7.5 
With the assumption of having enough registers for 
all threads in a SM (which is true).
Because each Streaming Multiprocessor (SM)
can have a maximum of 16 blocks and 1024 threads,
and each block can have a maximum of 1024 threads, 
I set the number of threads per block to 64 to occupy 
the maximum number of blocks and threads per SM.
64*16=1024
*/
#define MAX_DIMBLOCK 64


int print_bellmanFord_sequential(int , int * , int * , int );
void printPath(int* , int , int );
int print_bellmanFord_sequential_txt(char* , int , int* , int* , int );
void printPath_txt(FILE*, int* , int , int );
int print_bellmanFord_negativeCicle(char*);
char * filenameInizialzie(char[], int );
int adjacencyMatrixGen1D(int, int , int  , float , unsigned int , int *);
int printGraph1D(int * , int );
int bellmanFord_cuda(int *  , int ,  int , int *, int* , dim3  , dim3 , int );
int readGraphFromFile1D(int *, int , char *);
int printCSV(const char [],int , int , int , int , double , double , double , double );
double timeval_to_microseconds(struct timeval *, struct timeval *);



#define TESTING
#define SHARED


//We use a structure to work with edges so that we can pass the edges to the kernel

/// @brief structure used to define an arch
typedef struct {
    int start;
    int end;
    int weight;
} Edge;


/// @brief variable to indicate to the main that the distance vector has been updated during the iteration
__device__ int Updated;

/// @brief Kernel without the use of shared memory
/// @param edgeNumber Number of edges (excluding infinite weight)
/// @param device_EdgeArray Pointer to the edges
/// @param device_Distances Pointer to the distances
/// @return none
__global__ void bellmanFord_kernel_noShared ( int edgeNumber, Edge * device_EdgeArray, int * device_Distances) {
    
    int oldval, newval;
    unsigned int global_Id = blockIdx.x * blockDim.x + threadIdx.x;

    Updated=0;

    const unsigned int  i = device_EdgeArray[global_Id].start;
    const unsigned int  j = device_EdgeArray[global_Id].end;

    if(device_Distances[i]!=INF && global_Id < edgeNumber){
        if( device_Distances[j]> device_Distances[i] + device_EdgeArray[global_Id].weight ) {
                
            Updated=1;

            do { //for device_Distances
                oldval = device_Distances[j];
                newval=  device_Distances[i] + device_EdgeArray[global_Id].weight;
                newval=fminf(oldval, newval);

            } while( (atomicCAS((int*)&device_Distances[j], oldval, newval) != newval) );
        }
    }
}



/// @brief Kernel with the use of shared memory for the edge
/// @param edgeNumber same
/// @param device_EdgeArray  same
/// @param device_Distances same
/// @return none
__global__ void bellmanFord_kernel ( int edgeNumber, Edge * device_EdgeArray, int * device_Distances) {
    
    int oldval, newval;
    unsigned int global_Id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int local_Id = threadIdx.x;

    Updated=0;

    //In this case, shared memory is used as a buffer 
    //and not to share data between threads within the same block
    __shared__ Edge buffer[MAX_DIMBLOCK];


    if(global_Id < edgeNumber) {
        buffer[local_Id] = device_EdgeArray[global_Id];
    }
    

    __syncthreads();

    const unsigned int  i = buffer[local_Id].start;
    const unsigned int  j = buffer[local_Id].end;


    if(device_Distances[i]!=INF && global_Id < edgeNumber){
        if( device_Distances[j]> device_Distances[i] + buffer[local_Id].weight ) {
               
            Updated=1;

            do { //for device_Distances
                oldval = device_Distances[j];
                newval= device_Distances[i] + buffer[local_Id].weight;
                newval=fminf(oldval, newval);
            } while( (atomicCAS((int*)&device_Distances[j], oldval, newval) != newval) );
        }
    }
}
 

/// @brief "Bellman-Ford function that invokes the kernel
/// @param host_Graph Adjacency matrix
/// @param V Number of vertices
/// @param src Starting vertex
/// @param host_Distances Vector of distances
/// @param blocksize Number of threads in a block
/// @param totalThread Total threads executed
/// @param tipe 0 = shared memory and L1 < sh; 1 = no shared memory and  L1 > sh; 2 = shared memory and L1 >sh
/// @return If errors have occurred
int bellmanFord_cuda(int * host_Graph , int V,  int src, int *host_Distances, int blocksize, int*  totalThread, int tipe) {

    hipGetLastError();

    int host_hasNegativeCycle = 0;
    int i,j;
    Edge* device_EdgeArray;
    int *device_Distances;
    Edge* host_EdgeArray;

    //OpenMP!
    int edgeNumber = 0; 
    #pragma omp parallel for default(none) shared(host_Graph,V) reduction(+:edgeNumber) private (i,j)
    for( i=0; i<V; i++){
        for( j=0; j<V; j++){ 
            if(host_Graph[i*V+j]!=INF){
                edgeNumber++; //We count the number of edges actually present in the adjacency matrix
            }
        }
    }


    int size_Distances = V * sizeof(int);
    int size_Graph = edgeNumber * sizeof(Edge);
    
    //Allocation of the edges vector
    host_EdgeArray = (Edge*)malloc(edgeNumber* sizeof(Edge));
    if (host_EdgeArray == NULL) {
        fprintf(stderr, "edge malloc error\n");
        return -1; 
    }


    //Populating the edges vector
    int x = 0;
    for( i=0; i<V; i++){
        for( j=0; j<V; j++){ 
            if(host_Graph[i*V+j]!=INF){
                host_EdgeArray[x].weight=host_Graph[i*V+j];
                host_EdgeArray[x].start=i;
                host_EdgeArray[x].end=j;   
                x++;            
            }
        }
    }



    //OpenMP!
    #pragma omp parallel for default(none) shared(host_Distances,V) private(i)
    for(i=0; i<V; i++) {
        host_Distances[i] = INF;
    }
    host_Distances[src] = 0;

    

    dim3 threadForBlocks(blocksize);
    dim3 blocksForGrid(((edgeNumber-1) / blocksize)+1); //we allocate a sufficient number of blocks so that each thread can work with an edge
    *totalThread = (threadForBlocks.x)*(blocksForGrid.x);


    //CUDA! maclloc
    CUDA_CHECK( hipMalloc((void**)&device_Distances, size_Distances) );
    CUDA_CHECK( hipMalloc((void**)&device_EdgeArray, size_Graph) );
    
    //copy
    CUDA_CHECK( hipMemcpy(device_EdgeArray, host_EdgeArray, size_Graph, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(device_Distances, host_Distances, size_Distances, hipMemcpyHostToDevice) );

    
    int iterNumber=0;
    int host_updated=0;

    if (tipe==0){ //ShMem > L1; yes shared
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(bellmanFord_kernel), hipFuncCachePreferShared); 
        for(int i=0; i<V-1; i++){//V-1 kernell calling
            bellmanFord_kernel <<< blocksForGrid, threadForBlocks >>> (edgeNumber, device_EdgeArray, device_Distances);
            
            CUDA_CHECK( hipDeviceSynchronize() );
            iterNumber++;
            hipMemcpyFromSymbol(&host_updated, HIP_SYMBOL(Updated), sizeof(int));
            if(host_updated==0){
                break;
            }

        }
    }else if(tipe==1){ //ShMem < L1; No shared
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(bellmanFord_kernel), hipFuncCachePreferL1); 
        for(int i=0; i<V-1; i++){//V-1 kernell calling
            bellmanFord_kernel_noShared <<< blocksForGrid, threadForBlocks >>> (edgeNumber, device_EdgeArray, device_Distances);
            
            CUDA_CHECK( hipDeviceSynchronize() );
            iterNumber++;
            hipMemcpyFromSymbol(&host_updated, HIP_SYMBOL(Updated), sizeof(int));
            if(host_updated==0){
                break;
            }
        }
    }else if(tipe==2){ //ShMem < L1; yes shared
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(bellmanFord_kernel), hipFuncCachePreferL1); 
        for(int i=0; i<V-1; i++){//V-1 kernell calling
           bellmanFord_kernel <<< blocksForGrid, threadForBlocks >>> (edgeNumber, device_EdgeArray, device_Distances);
            
            CUDA_CHECK( hipDeviceSynchronize() );
            iterNumber++;
            hipMemcpyFromSymbol(&host_updated, HIP_SYMBOL(Updated), sizeof(int));
            if(host_updated==0){
                break;
            }
        }
    }else{
        fprintf(stderr, "cuda tipe not allowed\n");
        return -1;
    }
    
    

    //Similarly, for the distance search, I opted not to use the define for improved code readability.
    if(iterNumber==V-1){
        host_hasNegativeCycle = 1;
        if (tipe==0){ 
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(bellmanFord_kernel), hipFuncCachePreferShared); 
                bellmanFord_kernel <<< blocksForGrid, threadForBlocks >>> (edgeNumber, device_EdgeArray, device_Distances);
                
                CUDA_CHECK( hipDeviceSynchronize() );
                iterNumber++;
                hipMemcpyFromSymbol(&host_updated, HIP_SYMBOL(Updated), sizeof(int));
                if(host_updated==0){
                    host_hasNegativeCycle=0;
                }
        }else if(tipe==1){ 
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(bellmanFord_kernel), hipFuncCachePreferL1); 
                bellmanFord_kernel_noShared <<< blocksForGrid, threadForBlocks >>> (edgeNumber, device_EdgeArray, device_Distances);
                
                CUDA_CHECK( hipDeviceSynchronize() );
                iterNumber++;
                hipMemcpyFromSymbol(&host_updated, HIP_SYMBOL(Updated), sizeof(int));
                if(host_updated==0){
                    host_hasNegativeCycle=0;
                }
        }else if(tipe==2){ //ShMem < L1; yes shared
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(bellmanFord_kernel), hipFuncCachePreferL1); 
            bellmanFord_kernel <<< blocksForGrid, threadForBlocks >>> (edgeNumber, device_EdgeArray, device_Distances);
                
                CUDA_CHECK( hipDeviceSynchronize() );
                iterNumber++;
                hipMemcpyFromSymbol(&host_updated, HIP_SYMBOL(Updated), sizeof(int));
                if(host_updated==0){
                    host_hasNegativeCycle=0;
                }
        }
    }



    // copy-back of the result
    if(host_hasNegativeCycle==0)
    CUDA_CHECK( hipMemcpy(host_Distances, device_Distances, size_Distances, hipMemcpyDeviceToHost) );





    // deallocation
    CUDA_CHECK( hipFree(device_EdgeArray) );
    CUDA_CHECK( hipFree(device_Distances) );

    free(host_EdgeArray);

    return host_hasNegativeCycle;
}






/// @brief 
/// @param argv[0] -> The executable's name includes the optimization with which it was compiled  
/// @param argv[1] -> number of vertices
/// @param argv[2] -> number of OpenMP threads 
/// @param argv[3] -> cuda tipe : 0 = shared memory L1 < sh; 1 = no shared memory L1 > sh; 2 = shared memory L1 >sh
/// @param argv[4] -> vertexs source number
/// @param argv[5] -> filename = File from which to read the matrix
/// @param argv[6] -> filename2 = File to write the results to
/// @return 
int main(int argc, char *argv[]){

    //OpenMP
    int num_threads = 1;
    int tipe = 0;
    //---

    //For testing
    struct timeval total_time_Start, total_time_End; 
    struct timeval algorithm_Start, algorithm_End; 
    struct timeval input_time_Start, input_time_End;
    struct timeval output_time_Start, output_time_End;
    double total_time; 
    double input_time; 
    double algorithm ; 
    double output_time;
    int opt = -1;
    //---

    //for BF
    char fileName[50];
    char fileName2[50];
    int V = DEFAULT_V; 
    int * graph1D;
    int sourceVertex = DEFAULT_SOURCE;
    int * dist; // Minimum distance from the source to each vertex
    int negativeCicle = -1;
    //---


    //TIME
    gettimeofday(&total_time_Start,NULL);



    //Arg
    for (int i = strlen(argv[0]); argv[0][i] != 'n'; i--) { //'n' from main
        if (isdigit(argv[0][i])) {
            opt = atoi(&argv[0][i]);
            break;  
        }
    }
    if(opt==-1){
        fprintf(stderr, "name of the program:%s, and opt: %d",argv[0],opt);
        exit(EXIT_FAILURE);
    }

    if(argc > 1){
        if (sscanf(argv[1], "%u", &V) == 1) {
        } else {
            fprintf(stderr, "vertexs number (1° arg) expected as number, instead it was \"%s\"\n", argv[1]);
            exit(EXIT_FAILURE);
        }
    }

    if(argc > 2){
        if (sscanf(argv[2], "%d", &num_threads) == 1) {
            //OpenMP
            omp_set_num_threads(num_threads);
            //---
        } else {
            fprintf(stderr, "number of threads (2° arg) expected as number, instead it was \"%s\"\n", argv[2]);
            exit(EXIT_FAILURE);
        }
    }

    if(argc > 3){
        if (sscanf(argv[3], "%d", &tipe) == 1) {
        } else {
            fprintf(stderr, "cuda tipe (3° arg) expected as number, instead it was \"%s\"\n", argv[3]);
            exit(EXIT_FAILURE);
        }
    }

    if(argc > 4){
        if (sscanf(argv[4], "%u", &sourceVertex) == 1) {
        } else {
            fprintf(stderr, "vertexs source number (4° arg) expected as number, instead it was \"%s\"\n", argv[4]);
            exit(EXIT_FAILURE);
        }
    }

    if(argc > 5){
        strcpy(fileName, argv[5]);
    }
    else{
        strcpy(fileName, DEFAULT_FILENAME1D);
    }         

    if(argc > 6){
        strcpy(fileName2, argv[6]);
    }
    else{
        strcpy(fileName2, DEFAULT_FILENAME_BellmanFord_Cuda);
    }



//-------------------------------------
    
      
 
   
    int blocksize = MAX_DIMBLOCK;


    //TIME
    gettimeofday(&input_time_Start,NULL);


    graph1D=(int*) malloc(V*V*sizeof(int));
    if(graph1D== NULL){
        fprintf(stderr,"malloc error");
    }

    
    //Reading the binary file without data partitioning
    if(readGraphFromFile1D(graph1D, V, fileName)!=0){
        fprintf(stderr,"reading file error");
        exit(EXIT_FAILURE); 
    }


    //TIME
    gettimeofday(&input_time_End,NULL); 
    



/*BellmanFord------------------------------------------------*/
    
    //TIME
    gettimeofday(&algorithm_Start,NULL);



    //BellmanFord
    dist = (int*)malloc(V * sizeof(int));
    if(dist == NULL){
        fprintf(stderr,"malloc error\n");
        exit(EXIT_FAILURE);
    }



 
    //calling bellmanFord
    int totalThread=0;
    negativeCicle = bellmanFord_cuda(graph1D , V,  sourceVertex, dist, blocksize, &totalThread, tipe );




    gettimeofday(&algorithm_End,NULL);
 


/*print BellmanFord----------------------------------------*/

    

    gettimeofday(&output_time_Start,NULL);//TIME



    if(negativeCicle == 0){
        print_bellmanFord_sequential_txt(fileName2,V, dist, NULL, sourceVertex);
    }else if(negativeCicle == 1){
        print_bellmanFord_negativeCicle(fileName2);
    }else{
        fprintf(stderr,"BellmanFord error\n");
    }




    gettimeofday(&output_time_End,NULL); //TIME



/*ending-----------------------------*/

    //free
    free(graph1D);
    free(dist);



    gettimeofday(&total_time_End,NULL); //TIME
        

    total_time = timeval_to_microseconds(&total_time_Start, &total_time_End); 
    input_time = timeval_to_microseconds(&input_time_Start, &input_time_End); 
    algorithm = timeval_to_microseconds(&algorithm_Start, &algorithm_End); 
    output_time = timeval_to_microseconds(&output_time_Start, &output_time_End);


    char risultato[30];
    sprintf(risultato, "%s_%d", "OMP+CUDA", tipe);
    if(printCSV(risultato, V, opt, num_threads, totalThread, total_time, algorithm, input_time, output_time)!=0){
        fprintf(stderr,"CSV error");
        exit(EXIT_FAILURE);
    }

    return 0;

    

}











/////////////////////////////////////////////////////////////////////////
int print_bellmanFord_sequential_txt(char* fileName, int V, int* dist, int* parent, int sourceVertex) {
    char *filenameV = (char *)malloc(strlen(fileName) + sizeof(char)*5);
    strcpy(filenameV, fileName);
    char stringV[10];
    sprintf(stringV, "%d", V);
    strcat(filenameV, stringV);

    char *filename_with_extension = (char *)malloc(strlen(filenameV) + sizeof(char)*5);
    strcpy(filename_with_extension, filenameV);
    strcat(filename_with_extension, ".txt");



    FILE* file = fopen(filename_with_extension, "w");
    if (file == NULL) {
        fprintf(stderr,"file error");
        return -1;
    }

    
    int i;
    for (i = 0; i < V; i++) {
        fprintf(file, "Minimum distance from source to vertex %d: %d\t", i, dist[i]);
        fprintf(file, "\n");
    }

    // Chiudi il file
    fclose(file);
    return 0;
}




///////////////////////////////////////////////////////////////////////////////////////////
int adjacencyMatrixGen1D(int V, int weightMax, int weightMin , float edgeProbability, unsigned int seed, int *graph) {
    
    int weight, i, j, temp;

/*--------------------------------------------------------------------*/

    //Graph generator
    srand(seed);
    
    for (i = 0; i < V; i++) {
        for (j = 0; j < V; j++) {
        
            if (((double)rand() / RAND_MAX) < edgeProbability) {
                weight = rand() % (weightMax - weightMin + 1) + weightMin;


                //we eliminate the obvious negative cycles (not guarantee the absence of negative cycles)
                if(weight < 0){

                    // We avoid the creation of a negative cycle originating from the same vertex. 
                    if(i == j){ 
                        weight = abs(weight);
                    }

                    // We avoid the creation of a negative cycle between two nodes
                    else if(graph[j*V+i] != ABSENTE_EDGE && graph[j*V+i] + weight <0 ){
                        temp = abs(graph[j*V+i]) + 1;
                        if(weightMax > temp )
                            weight = temp;
                        else
                            weight = weightMax;
                    }

                    
                        
                }


                graph[i*V+j] = weight;
                

            } else {
                graph[i*V+j] = ABSENTE_EDGE;
            }
            
        }
    }

    return 0;

}

///////////////////////////////////////////////////////////////////////////////////////////
int printGraph1D(int * graph, int V) {
    int x=ABSENTE_EDGE;
    for (int i = 0; i < V; i++) {
        for (int j = 0; j < V; j++) {
            x=graph[i*V+j];
            if(x==ABSENTE_EDGE) 
                printf("x\t");
            else 
                printf("%d\t", x);
        }
        printf("\n");
    }
    return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////
char * filenameInizialzie(char fileName [], int V){
    char *filenameV = (char *)malloc(strlen(fileName) + sizeof(char)*5);
    if(filenameV==NULL){
        fprintf(stderr,"malloc error\n");
        exit(EXIT_FAILURE);
    }
    strcpy(filenameV, fileName);
    char stringV[10];
    sprintf(stringV, "%d", V);
    strcat(filenameV, stringV);

    char *filename_with_extension = (char *)malloc(strlen(filenameV) + sizeof(char)*5);
    if(filename_with_extension==NULL){
        fprintf(stderr,"malloc error\n");
        exit(EXIT_FAILURE);
    }
    strcpy(filename_with_extension, filenameV);
    strcat(filename_with_extension, ".bin");
    free(filenameV);
    return filename_with_extension;
}

///////////////////////////////////////////////////////////////////////////////////////////
int print_bellmanFord_negativeCicle(char* fileName){
    char *filename_with_extension = (char *)malloc(strlen(fileName) + sizeof(char)*5);
    strcpy(filename_with_extension, fileName);
    strcat(filename_with_extension, ".txt");

    FILE* file = fopen(fileName, "w");
    if (file == NULL) {
        fprintf(stderr,"file error\n");
        return -1;
    }

    fprintf(file, "The graph contains a negative cycle\n");

    fclose(file);
    return 0;
}

/////////////////////////////////////////////////////////////////////////
int readGraphFromFile1D(int *graph, int V, char *filename) {
    char *filenameV = (char *)malloc(strlen(filename) + sizeof(char)*5);
    if(filenameV == NULL){
        fprintf(stderr, "malloc file error");
        return -1;
    }
    strcpy(filenameV, filename);
    char stringV[10];
    sprintf(stringV, "%d", V);
    strcat(filenameV, stringV);

    char *filename_with_extension = (char *)malloc(strlen(filenameV) + sizeof(char)*5);
    if(filename_with_extension == NULL){
        fprintf(stderr, "malloc file error");
        return -1;
    }
    strcpy(filename_with_extension, filenameV);
    strcat(filename_with_extension, ".bin");

    FILE *file = fopen(filename_with_extension, "rb");

    if (file == NULL) {
        fprintf(stderr, "File %s could not be opened for reading.\n", filename_with_extension);
        return -1;
    }


    if(fread(graph, sizeof(int), V*V, file) != V*V){
        fprintf(stderr, "Error reading binary file.\n");
        fclose(file);
        return -1;
    }


    free(filename_with_extension);
    free(filenameV);
    fclose(file); 
    return 0;
}

/////////////////////////////////////////////////////////////////////////
double timeval_to_microseconds(struct timeval *time1, struct timeval *time2) {
    unsigned long long temp1 = (unsigned long long)time1->tv_sec * 1000000 + time1->tv_usec;
    unsigned long long temp2 = (unsigned long long)time2->tv_sec * 1000000 + time2->tv_usec;

    return (double)(temp2 - temp1)/(double)1000000;
}

/////////////////////////////////////////////////////////////////////////
int printCSV(const char tipe[],int V, int opt, int numThreads, int size, double total_time, double algorithm, double input_time, double output_time){
    
    
    FILE *fp2; 
    char path2[200];

    if(strcmp(tipe, "OMP+MPI") == 0){
        sprintf(path2, "Informations/OMP_MPI/opt%d/%dvertices.csv", opt, V); 
    }else if(strcmp(tipe, "OMP+CUDA_0") == 0 ||strcmp(tipe, "OMP+CUDA_1") == 0 || strcmp(tipe, "OMP+CUDA_2") == 0 ){
        sprintf(path2, "Informations/OMP_CUDA/opt%d/%dvertices.csv", opt, V);   
    }else if(strcmp(tipe, "none") == 0){
        sprintf(path2, "Informations/SEQUENTIAL/opt%d/%dvertices.csv", opt, V);
    }else{
         fprintf(stderr, "Unexpected compilation type: %s\n", tipe);
         return -2;
    }
    char *filename2 = path2;
    fp2 = fopen(filename2, "a+");
    if (fp2 == NULL) {
        fprintf(stderr, "Unable to open file: %s\n", filename2);
        return -1;
    }
    fprintf(fp2, "%s;%d;%d;%06f;%06f;%06f;%06f;\n", tipe, numThreads, size, total_time, algorithm, input_time, output_time);
    fclose(fp2);
    return 0;
}



